#include "Calculator.h"

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <stdexcept>
#include <Windows.h>
#include "common.h"

CCalculator::CCalculator(int width, int height, float FOV)
	: m_width(width), m_height(height), m_FOV(FOV)
{
	hipError_t cudaStatus;

	cudaStatus = hipMallocPitch((void**)&m_d_buffer, &m_pitch, width * sizeof(float), height);

	if (cudaStatus != hipSuccess) {
		ReactToCudaError(cudaStatus);
	}

}

CCalculator::~CCalculator()
{
	if (m_d_buffer)
	{
		hipError_t status = hipFree(m_d_buffer);
		if (status != hipSuccess)
		{
			ReactToCudaError(status);
		}
	}
}
