#include "hip/hip_runtime.h"
#include "TextureFiller.h"

#include "TextureInfo.h"

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <stdexcept>
#include <Windows.h>
#include "common.h"

#include "assert.h"

CTextureFiller::CTextureFiller(int width, int height, float FOV)
	: m_width(width), m_height(height),
	m_fL(height / 2.0f / tanf(FOV / 2.0f))
{
	hipError_t cudaStatus;

	cudaStatus = hipMallocPitch((void**)&m_d_buffer, &m_pitch, width * sizeof(float), height);

	if (cudaStatus != hipSuccess) {
		ReactToCudaError(cudaStatus);
	}

}

CTextureFiller::~CTextureFiller()
{
	if (m_d_buffer)
	{
		hipError_t status = hipFree(m_d_buffer);
		if (status != hipSuccess)
		{
			ReactToCudaError(status);
		}
	}
}

void CTextureFiller::FillTexture(CTextureInfo& tex)
{
	assert(tex.GetElementSize() == sizeof(float));
	hipError_t status;

	hipArray *cuArray;

	hipGraphicsResource* resources[] = { tex.GetCudaResource() };
	status = hipGraphicsMapResources(1, resources);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

	status = hipGraphicsSubResourceGetMappedArray(&cuArray, tex.GetCudaResource(), 0, 0);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

	status = hipMemcpy2DToArray(cuArray, 0, 0, GetBuffer(), GetPitch(), GetWidth() * sizeof(float), GetHeight(), hipMemcpyDeviceToDevice);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

	status = hipGraphicsUnmapResources(1, resources);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

}

void CTextureFiller::UpdateBuffer(const FrameParameters &params)
{
	dim3 Db = dim3(8, 8);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((GetWidth() + Db.x - 1) / Db.x, (GetHeight() + Db.y - 1) / Db.y);

	KernelParameters kParams;
	kParams.tFrameParams = params;
	kParams.width = GetWidth();
	kParams.height = GetHeight();
	kParams.L = GetL();
	kParams.pitch = GetPitch();

	LaunchKernel(kParams);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		ReactToCudaError(err);
	}
}
