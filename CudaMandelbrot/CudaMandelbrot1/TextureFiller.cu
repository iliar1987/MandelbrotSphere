#include "TextureFiller.h"

#include "TextureInfo.h"

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <stdexcept>
#include <Windows.h>
#include "common.h"

#include "assert.h"

CTextureFiller::CTextureFiller(int width, int height, float FOV)
	: m_width(width), m_height(height), m_FOV(FOV)
{
	hipError_t cudaStatus;

	cudaStatus = hipMallocPitch((void**)&m_d_buffer, &m_pitch, width * sizeof(float), height);

	if (cudaStatus != hipSuccess) {
		ReactToCudaError(cudaStatus);
	}

}

CTextureFiller::~CTextureFiller()
{
	if (m_d_buffer)
	{
		hipError_t status = hipFree(m_d_buffer);
		if (status != hipSuccess)
		{
			ReactToCudaError(status);
		}
	}
}

void CTextureFiller::FillTexture(CTextureInfo& tex)
{
	assert(tex.GetElementSize() == sizeof(float));
	hipError_t status;

	hipArray *cuArray;

	hipGraphicsResource* resources[] = { tex.GetCudaResource() };
	status = hipGraphicsMapResources(1, resources);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

	status = hipGraphicsSubResourceGetMappedArray(&cuArray, tex.GetCudaResource(), 0, 0);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

	status = hipMemcpy2DToArray(cuArray, 0, 0, GetBuffer(), GetPitch(), GetWidth() * sizeof(float), GetHeight(), hipMemcpyDeviceToDevice);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

	status = hipGraphicsUnmapResources(1, resources);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

}
