#include "hip/hip_runtime.h"
#include "TextureFiller.h"

#include "TextureInfo.h"

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <stdexcept>
#include <Windows.h>
#include "common.h"

#include "assert.h"

#include "..\HighPrecision1\FP128.cuh"

CTextureFiller::CTextureFiller(int width, int height, float FOV)
	: m_width(width), m_height(height),
	m_fL(height / 2.0f / tanf(FOV / 2.0f))
{
	hipError_t cudaStatus;

	cudaStatus = hipMallocPitch((void**)&m_d_buffer, &m_pitch, width * sizeof(float), height);

	if (cudaStatus != hipSuccess) {
		ReactToCudaError(cudaStatus);
	}

	m_poleCoords.x = new CFixedPoint128{ 0,0 };
	m_poleCoords.y = new CFixedPoint128{ 0,0 };
}

CTextureFiller::~CTextureFiller()
{
	delete m_poleCoords.x;
	delete m_poleCoords.y;
	if (m_d_buffer)
	{
		hipError_t status = hipFree(m_d_buffer);
		if (status != hipSuccess)
		{
			ReactToCudaError(status);
		}
	}
}

void CTextureFiller::FillTexture(CTextureInfo& tex)
{
	assert(tex.GetElementSize() == sizeof(float));
	hipError_t status;

	hipArray *cuArray;

	hipGraphicsResource* resources[] = { tex.GetCudaResource() };
	status = hipGraphicsMapResources(1, resources);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

	status = hipGraphicsSubResourceGetMappedArray(&cuArray, tex.GetCudaResource(), 0, 0);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

	status = hipMemcpy2DToArray(cuArray, 0, 0, GetBuffer(), GetPitch(), GetWidth() * sizeof(float), GetHeight(), hipMemcpyDeviceToDevice);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

	status = hipGraphicsUnmapResources(1, resources);
	if (status != hipSuccess)
	{
		ReactToCudaError(status);
	}

}

void CTextureFiller::UpdateBuffer(const FrameParameters &params)
{
	KernelParameters kParams;
	kParams.tFrameParams = params;
	kParams.width = GetWidth();
	kParams.height = GetHeight();
	kParams.L = GetL();
	kParams.pitch = GetPitch();

	LaunchKernel(kParams);
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		ReactToCudaError(err);
	}
}

void CTextureFiller::PoleCoordsGet(float& x, float &y)
{
	x = static_cast<float>(*m_poleCoords.x);
	y = static_cast<float>(*m_poleCoords.y);
}

void CTextureFiller::PoleCoordsAdd(float dx, float dy)
{
	*m_poleCoords.x += CFixedPoint128(dx);
	*m_poleCoords.y += CFixedPoint128(dy);
}

void CTextureFiller::PoleCoordsSet(float x, float y)
{
	*m_poleCoords.x = CFixedPoint128(x);
	*m_poleCoords.y = CFixedPoint128(y);
}

void CTextureFiller::PoleCoordsZoom(float3 vForward, float rho, float rho_new)
{
	float temp = sqrtf(vForward.x * vForward.x + vForward.y * vForward.y);
	float theta = atan2f(temp, vForward.z);
	float phi = atan2f(vForward.y, vForward.x);

	float rho_delta = rho_new - rho;

	float dr = 2 * rho_delta * tanf(theta / 2);
	float dx = dr * cosf(phi);
	float dy = dr * sinf(phi);

	*m_poleCoords.x += CFixedPoint128(-dx);
	*m_poleCoords.y += CFixedPoint128(-dy);
}
