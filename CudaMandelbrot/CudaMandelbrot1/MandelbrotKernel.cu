#include "hip/hip_runtime.h"
#include "MandelbrotKernel.h"

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <stdexcept>
#include <Windows.h>
#include "common.h"

#include "utilities.cuh"

#include "../HighPrecision1/FP128.cuh"

#define OUT
#define IN

typedef void(Func_GetThetaPhi)(OUT float&, OUT float&, int, int, const CTextureFiller::KernelParameters&);

template<Func_GetThetaPhi GetThetaPhi>
__global__ void kernMandelbrot(float* buffer, CTextureFiller::KernelParameters params,CFixedPoint128 xPole,CFixedPoint128 yPole)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	//if (params.tFrameParams.t > 10.0 && x == 1000 && y == 700)
	//{
	//	asm("brkpt;");
	//}

	if (x >= params.width || y >= params.height) return;

	float *pixel = buffer + (y * params.width + x);

	float theta, phi;
	GetThetaPhi(theta, phi, x, y, params);

	float R = params.tFrameParams.rho * tanf( theta / 2.0f) * 2.0f;
	float fX = R*cosf(phi);
	float fY = R*sinf(phi);
	//*pixel = sqrtf(fY*fY + fX*fX);
	//*pixel = theta / PIf;
	//*pixel = powf(sin(theta), 3) * (cosf(3.0f*(phi + params.tFrameParams.t))) * 0.5f + 0.5f;


	if (fX*fX + fY*fY >= 4.0f)
	{
		*pixel = 0.0f;
	}
	else
	{
		CComplexFP128 c(fX, fY);
		c.x += xPole;
		c.y += yPole;
		CComplexFP128 z(c);
		int i = 0;

		while (i < params.tFrameParams.nIterations
			&& !((z.x.hihi & 0x80000000) >> 1 != z.x.hihi & 0x40000000)
			&& !((z.y.hihi & 0x80000000) >> 1 != z.y.hihi & 0x40000000) )
		{
			CFixedPoint128 z_x_sqr = z.x.Sqr();
			CFixedPoint128 z_y_sqr = z.y.Sqr();

			CFixedPoint128 sumOfSquares(z_x_sqr);
			sumOfSquares += z_y_sqr;
			if (sumOfSquares.IsNeg())
				break;

			z.y = z.x * z.y;
			z.y <<= 1;

			z.x = z_x_sqr;
			z_y_sqr.Negate();
			z.x = z_y_sqr;

			z += c;
			++i;
		}
		*pixel = (float)i;

		//z = z.Sqr();
		
		//float fa = (float)(z.x);
		//float fb = (float)(z.y);
		////*pixel = fa*fa + fb*fb;
		////z.Sqr();
		//*pixel = atan2f((float)z.y, (float)z.x);
	}
	
}

void CMandelbrotTextureFiller::LaunchKernel(const KernelParameters& params)
{
	dim3 Db = dim3(8, 8);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((params.width + Db.x - 1) / Db.x, (params.height + Db.y - 1) / Db.y);

	kernMandelbrot<GetThetaPhiSpherical> <<< Dg, Db >>> (GetBuffer(), params,*m_poleCoords.x,*m_poleCoords.y);
}

CMandelbrotTextureFiller::CMandelbrotTextureFiller(int width, int height, float FOV)
	: CTextureFiller(width, height, FOV)
{
	m_poleCoords.x = new CFixedPoint128 { 0,0 };
	m_poleCoords.y = new CFixedPoint128{ 0,0 };
}

CMandelbrotTextureFiller::~CMandelbrotTextureFiller()
{
	delete m_poleCoords.x;
	delete m_poleCoords.y;
}


void CMandelbrotTextureFiller::PoleCoordsGet(float& x, float &y)
{
	x = static_cast<float>(*m_poleCoords.x);
	y = static_cast<float>(*m_poleCoords.y);
}

void CMandelbrotTextureFiller::PoleCoordsAdd(float dx, float dy)
{
	*m_poleCoords.x += CFixedPoint128(dx);
	*m_poleCoords.y += CFixedPoint128(dy);
}

void CMandelbrotTextureFiller::PoleCoordsSet(float x, float y)
{
	*m_poleCoords.x = CFixedPoint128(x);
	*m_poleCoords.y = CFixedPoint128(y);
}

void CMandelbrotTextureFiller::PoleCoordsZoom(float3 vForward, float rho, float rho_new)
{
	float temp = sqrtf(vForward.x * vForward.x + vForward.y * vForward.y);
	float theta = atan2f(temp, vForward.z);
	float phi = atan2f(vForward.y, vForward.x);

	float rho_delta = rho_new - rho;

	float dr = 2 * rho_delta * tanf(theta / 2);
	float dx = dr * cosf(phi);
	float dy = dr * sinf(phi);

	*m_poleCoords.x += CFixedPoint128(-dx);
	*m_poleCoords.y += CFixedPoint128(-dy);
}

