
#include "MandelbrotKernel.h"

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <stdexcept>
#include <Windows.h>
#include "common.h"

#include "utilities.cuh"

#include "ComplexIterationKernel.cuh"

__device__ bool IterateMandelbrot(IN OUT CComplexFP128 &z, IN const CComplexFP128 &c)
{
	if (((z.x.hihi & 0x80000000) >> 1) != (z.x.hihi & 0x40000000)
		|| ((z.y.hihi & 0x80000000) >> 1) != (z.y.hihi & 0x40000000))
		return true;

	CFixedPoint128 z_x_sqr = z.x.Sqr();
	CFixedPoint128 z_y_sqr = z.y.Sqr();

	CFixedPoint128 sumOfSquares(z_x_sqr);
	sumOfSquares += z_y_sqr;
	if (sumOfSquares.IsNeg())
		return true;

	z.y = z.x * z.y;
	z.y <<= 1;

	z.x = z_x_sqr;
	z.x -= z_y_sqr;

	z += c;

	return false;
}

typedef CComplexIterationTextureFiller<IterateMandelbrot> CMandelbrotTextureFiller;

CTextureFiller * CreateMandelbrotTextureFiller(int width, int height, float FOV)
{
	return new CMandelbrotTextureFiller(width, height, FOV);
}
