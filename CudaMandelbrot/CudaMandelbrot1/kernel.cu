#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <stdexcept>
#include <Windows.h>
#include "common.h"

#define ARR_AS_FLOAT3(arr) (*((float3*) (arr) ))

union Quaternion
{
	struct
	{
		float i,j,k,r;
	};
	float arr[4];
	__host__ __device__ Quaternion(float4 vec)
		: r(vec.w), i(vec.x),j(vec.y),k(vec.z)
	{
	}
	__host__ __device__ Quaternion(float i, float j, float k, float r)
		: i(i),j(j),k(k),r(r)
	{

	}

	__host__ __device__ Quaternion operator * (const Quaternion &q) const
	{
		return Quaternion(r * q.r		- i * q.i - j * q.j - k * q.k,
			r * q.i + i * q.r			+ j * q.k - k * q.j,
			r * q.j + j * q.r			+ k * q.i - i * q.k,
			r * q.k + k * q.r			+ i * q.j - j * q.i);
	}

	__host__ __device__ Quaternion operator + (const Quaternion &other) const
	{
		return Quaternion(r + other.r,
			i + other.i,
			j + other.j,
			k + other.k);
	}

	__host__ __device__ Quaternion Conj() const
	{
		return Quaternion(r, -i, -j, -k);
	}

	__host__ __device__ float3 RotateVector(float3 pos) const
	{
		Quaternion q(pos.x, pos.y, pos.z,0);
		q = operator*(q * Conj());
		return float3 { q.i, q.j, q.k };
	}
};


__host__ __device__ float3 operator * (float x, float3 v)
{
	return{ x*v.x, x*v.y, x*v.z };
}

__host__ __device__ float3 operator * (float3 v, float x)
{
	return{ x*v.x, x*v.y, x*v.z };
}

__host__ __device__ float3 operator + (float3 u, float3 v)
{
	return{ u.x + v.x,u.y + v.y,u.z + v.z };
}
	

__global__ void kernSpherical(float4* buffer, const int width, const int height, const size_t pitch, const float t, const float L, const float3 vCameraRight, const float3 vCameraUp, const float3 vCameraForward)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	float4 *pixel = buffer + (y * width + x);
	
	float3 posCamera = (float)(x - width / 2) * vCameraRight + (float)(y - height / 2) * vCameraUp + vCameraForward * (float)L;

	const float r = sqrtf(posCamera.x * posCamera.x + posCamera.y * posCamera.y);

	const float theta = atan2f(posCamera.z,r);
	const float phi = atan2f(posCamera.y, posCamera.x);
	
	pixel->x = (1.0f + cosf(phi))*0.5f;
	pixel->y = theta / PIf + 0.5f;
	pixel->z = (1.0f + sinf(phi))*0.5f;;
	pixel->w = 1.0f;
}



SimpleFillTexture::SimpleFillTexture(int width, int height, float FOV)
	: m_width(width), m_height(height), m_FOV(FOV)
{
	hipError_t cudaStatus;

	cudaStatus = hipMallocPitch((void**)&m_d_buffer, &m_pitch, width * sizeof(float4), height);

	if (cudaStatus != hipSuccess) {
		ReactToCudaError(cudaStatus);
	}

}

SimpleFillTexture::~SimpleFillTexture()
{
	if (m_d_buffer)
	{
		hipError_t status = hipFree(m_d_buffer);
		if (status != hipSuccess)
		{
			ReactToCudaError(status);
		}
	}
}



void SimpleFillTexture::UpdateBuffer(float vCamRight[3], float vCamUp[3], float vCamForward[3])
{
	static int dir = 1;
	static float t = 0;
	t += 0.02f * dir;
	if (t > 1)
	{
		t = 1;
		dir = -1;
	}
	else if (t < 0)
	{
		dir = 1;
		t = 0;
	}

	float L = (float)m_height / 2.0f / tanf(m_FOV / 2.0f);
	dim3 Db = dim3(8, 8);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((GetWidth() + Db.x - 1) / Db.x, (GetHeight() + Db.y - 1) / Db.y);

	kernSpherical << <Dg, Db >> > (m_d_buffer, GetWidth(), GetHeight(), GetPitch(), t,L, ARR_AS_FLOAT3( vCamRight), ARR_AS_FLOAT3( vCamUp), ARR_AS_FLOAT3( vCamForward));

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		ReactToCudaError(err);
	}
}