#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <utility>

#include <iostream>

#include "FP128.cuh"

__global__ void mulKernel(CFixedPoint128 *c, const CFixedPoint128 *a, const CFixedPoint128 *b)
{
    int i = threadIdx.x;
	c[i] = a[i] * b[i];
}


__global__ void shlKernel(CFixedPoint128 *c, const CFixedPoint128 *a, const CFixedPoint128 *b)
{
	int i = threadIdx.x;
	CFixedPoint128 temp = a[i];
	temp <<= 1;
	c[i] = temp;
}

__global__ void shrKernel(CFixedPoint128 *c, const CFixedPoint128 *a, const CFixedPoint128 *b)
{
	int i = threadIdx.x;
	CFixedPoint128 temp = a[i];
	temp >>= 1;
	c[i] = temp;
}

__global__ void divideByPow2Kernel(CFixedPoint128 *c, const CFixedPoint128 *a, const CFixedPoint128 *b)
{
	int i = threadIdx.x;
	CFixedPoint128 temp = a[i];
	temp >>= (i+5);
	c[i] = temp;
}


__global__ void isNegKernel(CFixedPoint128 *c, const CFixedPoint128 *a, const CFixedPoint128 *b)
{
	int i = threadIdx.x;
	c[i].hi = 0;
	if (a[i].IsNeg())
	{
		c[i].lo = 1;
	}
	else
	{
		c[i].lo = 0;
	}
}

__global__ void subtractKernel(CFixedPoint128 *c, const CFixedPoint128 *a, const CFixedPoint128 *b)
{
	int i = threadIdx.x;

	/*CFixedPoint128 x = a[i];
	x.Negate();
	x += b[i];
	c[i] = x;*/
	CFixedPoint128 x = b[i];
	x -= a[i];
	c[i] = x;
}

__global__ void ComplexSqrKernel(CComplexFP128 *c)
{
	int i = threadIdx.x;
	c[i] = c[i].Sqr();
}

__global__ void FromFloatKernel(CFixedPoint128* output, const float* input)
{
	output[threadIdx.x] = CFixedPoint128(input[threadIdx.x]);
}

__global__ void ToFloatKernel(float* output,const CFixedPoint128* input)
{
	output[threadIdx.x] = (float)input[threadIdx.x];
}


void PrintFromFloat(float f)
{
	CFixedPoint128 fp128(f);
	uint32_t ui32F = FloatToBin32(f);
	float fResult = (float)fp128;
	uint32_t ui32Result = FloatToBin32(fResult);
	printf("%08x = ", ui32F);
	std::cout << f << " = " << fp128 << " = " << fResult;
	printf(" = %08x\r\n", ui32Result);

}

void TestFromFloat()
{
	printf("Testing From float conversion: \r\n");
	float arr[] = { 0.25f,0.5f,1.0f,1.5f,1.25f,0.75f,-2.5f,-1.25f,-0.5f,-0.000000000123f,4123423e-30f,-1.0f,2.0f,-4.0f,0.0f};
	for (float x : arr)
	{
		PrintFromFloat(x);
	}
}

void TestFromFloatGPU()
{
	printf("Testing From float and back conversion (GPU): \r\n");
	float arr[] = { 0.25f,0.5f,1.0f,1.5f,1.25f,0.75f,-2.5f,-1.25f,-0.5f,-0.000000000123f,4123423e-30f,-1.0f,2.0f,-4.0f,0.0f };
	const int N = sizeof(arr) / sizeof(float);
	float* d_arr;
	float h_result[N];
	CFixedPoint128* d_fp128;
	CFixedPoint128 h_fp128[N];

	hipMalloc(&d_arr, N * sizeof(float));
	hipMemcpy(d_arr, arr, N * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);
	hipMalloc(&d_fp128, sizeof(CFixedPoint128)*N);

	FromFloatKernel <<< 1, N >>> (d_fp128, d_arr);
	ToFloatKernel << < 1, N >> > (d_arr, d_fp128);
	
	hipMemcpy(h_fp128, d_fp128, sizeof(CFixedPoint128) * N, hipMemcpyKind::hipMemcpyDeviceToHost);
	hipMemcpy(h_result, d_arr, sizeof(float) * N, hipMemcpyKind::hipMemcpyDeviceToHost);

	hipFree(d_arr);
	hipFree(d_fp128);

	for (int i = 0;i < N; ++i)
	{
		std::cout << arr[i] << " = " << h_fp128[i] << " = " << h_result[i] << std::endl;
	}
	

	//FromFloatKernel <<< dim3()
}

hipError_t TestComplex();
typedef void CudaOp(CFixedPoint128 *c, const CFixedPoint128 *a, const CFixedPoint128 *b);

hipError_t PerformOpWithCuda(CudaOp* op, CFixedPoint128 *c, const CFixedPoint128 *a, const CFixedPoint128 *b, unsigned int size);

int main()
{
	TestFromFloat();
	TestFromFloatGPU();

    const int arraySize = 3;
	const CFixedPoint128 a[arraySize] = { {0x1010101010101010L,0x1010101010101010L },{1,0} ,{ 0x2020202020202020L,0x4020202020202020L } };
	const CFixedPoint128 b[arraySize] = { { 0x3010101010101010L, 0x1010101010101010L },{ 1,0 } ,{ 0x2020202020202020L, 0x2020202020202020L } };
	CFixedPoint128 c[arraySize] = { {0,0},{1,1} };
	hipError_t cudaStatus;

    cudaStatus = PerformOpWithCuda(&mulKernel, c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mul failed!");
        return 1;
    }
	for (int i = 0;i < arraySize;++i)
	{

		std::cout << a[i] << "\t*\t" << b[i] << "\t=\t" << c[i] << std::endl;

	}

	cudaStatus = PerformOpWithCuda(&subtractKernel,c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "subtract failed!");
		return 1;
	}
	for (int i = 0;i < arraySize;++i)
	{
		std::cout << b[i] << "\t-\t" << a[i] << "\t=\t" << c[i] << std::endl;
	}

	memcpy((void*)a, (void*)c, arraySize * sizeof(CFixedPoint128));
	cudaStatus = PerformOpWithCuda(&isNegKernel, c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "isneg failed!");
		return 1;
	}
	for (int i = 0;i < arraySize;++i)
	{
		std::cout << a[i] << ".IsNeg() \t=\t" << c[i] << std::endl;
	}
	
	cudaStatus = PerformOpWithCuda(&shlKernel, c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "shl failed!");
		return 1;
	}
	for (int i = 0;i < arraySize;++i)
	{
		std::cout << a[i] << "<<1 \t=\t" << c[i] << std::endl;
	}

	cudaStatus = PerformOpWithCuda(&shrKernel, c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "shl failed!");
		return 1;
	}
	for (int i = 0;i < arraySize;++i)
	{
		std::cout << a[i] << ">>1 \t=\t" << c[i] << std::endl;
	}

	cudaStatus = PerformOpWithCuda(&divideByPow2Kernel, c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "divide by pow 2 failed!");
		return 1;
	}
	for (int i = 0;i < arraySize;++i)
	{
		std::cout << a[i] << "/2**"<<i+5<<" \t=\t" << c[i] << std::endl;
	}

	cudaStatus = TestComplex();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "complex square");
		return 1;
	}

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t PerformOpWithCuda(CudaOp* op, CFixedPoint128 *c, const CFixedPoint128 *a, const CFixedPoint128 *b, unsigned int size)
{
	CFixedPoint128 *dev_a = 0;
	CFixedPoint128 *dev_b = 0;
	CFixedPoint128 *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(CFixedPoint128));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(CFixedPoint128));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(CFixedPoint128));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(CFixedPoint128), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(CFixedPoint128), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    (*op)<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(CFixedPoint128), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

hipError_t TestComplex()
{
	CComplexFP128 C[] = { { 1.5f,1.5f },{ 1.4f,1.4f } ,{ 1,0 },{ 0,1 },{ 0,-1 },{ 0.5f,-0.5f } };
	
	const unsigned int  size = sizeof(C) / sizeof(CComplexFP128);
	CComplexFP128 C_sqr[size];
	for (CComplexFP128 &c : C)
	{
		std::cout << "c = " << c << std::endl;
		std::cout << "|c| >= 2  =  " << c.OutsideRadius2() << std::endl;
	}
	CComplexFP128 *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(CComplexFP128));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_c, C, size * sizeof(CComplexFP128), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	ComplexSqrKernel <<<1, size >>>(dev_c);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(C, dev_c, size * sizeof(CComplexFP128), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	std::cout << "After complex square:" << std::endl;
	for (CComplexFP128 &c : C)
	{
		std::cout << "c = " << c << std::endl;
		std::cout << "|c| >= 2  =  " << c.OutsideRadius2() << std::endl;
	}

Error:
	hipFree(dev_c);
	return cudaStatus;
}

